#include "hip/hip_runtime.h"

#include "TargetValuesKernel.hpp"
#include "Constants.hpp"
#include "../Types.cuh"
#include <hip/hip_runtime.h>
#include <cassert>

using namespace rnn;
using namespace rnn::cuda;

__global__
void targetValuesKernel(CuMatrix nextTargetActivation, CuMatrix batchRewards,
                        float discountFactor, CuMatrix outTargetValue) {

  const unsigned batchIndex = blockIdx.x;

  float maxVal = *Elem(nextTargetActivation, batchIndex, 0);
  for (unsigned i = 1; i < nextTargetActivation.cols - 1; i++) {
    maxVal = fmaxf(maxVal, *Elem(nextTargetActivation, batchIndex, i)));
  }

  float target = *Elem(batchRewards, batchIndex, 0) + discountFactor * maxVal;

  // TODO: this is unnecessary as we only need to set the target on one output connection,
  // corresponding to the action actually performed.
  for (unsigned i = 0; i < outTargetValue.cols; i++) {
    *Elem(outTargetValue, batchIndex, i) = target;
  }
}

void TargetValuesKernel::Apply(CuMatrix nextTargetActivation, CuMatrix batchRewards,
                               float discountFactor, CuMatrix outTargetValue, hipStream_t stream) {

  assert(nextTargetActivation.activation.cols == outTargetValue.cols);
  assert(nextTargetActivation.activation.rows == outTargetValue.rows);
  assert(batchRewards.cols == 1);
  assert(batchRewards.rows == outTargetValue.rows);
  assert(discountFactor > 0.0f && discountFactor <= 1.0f);

  int tpb = 1;
  int bpg = outTargetValue.rows;

  targetValuesKernel<<<bpg, tpb, 0, stream>>>(
      nextTargetActivation, batchRewards, discountFactor, outTargetValue);
}
